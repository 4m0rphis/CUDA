#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <vector>
#include <chrono>

#define checkForCudaErr(value)                                    \
    {                                                             \
        hipError_t err = value;                                  \
        if (err != hipSuccess)                                   \
        {                                                         \
            fprintf(stderr, "Error %s at line %d in file %s\n",   \
                    hipGetErrorString(err), __LINE__, __FILE__); \
            exit(-1);                                             \
        }                                                         \
    }

__global__ void blurBox(unsigned char *rgb, unsigned char *out, int rows, int cols, int radius)
{

    if (radius % 2 == 0)
    {
        return;
    }

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    int row = i / cols;
    int col = i % cols;

    int limit = ((radius - 1) / 2) - 1;

    for (int channel = 0; channel < 3; channel++)
    {
        int sum = 0;
        for (int r = -limit - 1; r <= limit + 1; r++)
        {
            for (int c = -limit - 1; c <= limit + 1; c++)
            {
                sum += rgb[3 * ((row + r) * cols + (col + c)) + channel];
            }
        }
        out[3 * i + channel] = sum / (radius * radius);
    }
}

int main()
{
    cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED);
    auto rgb = m_in.data;
    auto width = m_in.cols;
    auto height = m_in.rows;
    unsigned char *d_rgb, *d_out;

    checkForCudaErr(hipMalloc(&d_rgb, width * height * 3));
    checkForCudaErr(hipMalloc(&d_out, width * height * 3));

    checkForCudaErr(hipMemcpy(d_rgb, rgb, width * height * 3, hipMemcpyHostToDevice));

    int block_size = 2;
    int grid_size = (width * height + block_size - 1) / block_size;

    hipEvent_t start, stop;

    checkForCudaErr(hipEventCreate(&start));
    checkForCudaErr(hipEventCreate(&stop));

    // Mesure du temps de calcul du kernel uniquement.
    checkForCudaErr(hipEventRecord(start));

    blurBox<<<grid_size, block_size>>>(d_rgb, d_out, height, width, 11);

    checkForCudaErr(hipEventRecord(stop));

    // hipDeviceSynchronize();

    std::vector<unsigned char> out(width * height * 3);
    checkForCudaErr(hipMemcpy(out.data(), d_out, width * height * 3, hipMemcpyDeviceToHost));

    checkForCudaErr(hipEventSynchronize(stop));
    float duration;
    checkForCudaErr(hipEventElapsedTime(&duration, start, stop));
    std::cout << "time=" << duration << std::endl;

    checkForCudaErr(hipEventDestroy(start));
    checkForCudaErr(hipEventDestroy(stop));

    cv::Mat m_out(height, width, CV_8UC3, out.data());
    cv::imwrite("out.jpg", m_out);

    hipFree(d_rgb);
    hipFree(d_out);

    return 0;
}